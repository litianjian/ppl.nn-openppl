#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include <vector>
#include <hip/hip_runtime.h>
#include <assert.h>

#include <fstream>
#include <sstream>
#include <string>
#include <iostream>
#include <unordered_map>
#include <hip/hiprtc.h>

#include "cudakernel/nn/conv/conv_fp16.h"
#include "cudakernel/nn/conv/gene_kernel.h"
#include "kernel_type.h"
#include "conv_common.h"
#include "common/init_lut.h"
#include "common/merge_split.h"

#include "ppl/nn/engines/cuda/module/cuda_compiler.h"
#include "ppl/nn/engines/cuda/module/cuda_module.h"

#include "float.h"

#define TIMES 4

#define SPK_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                                                                  \
    		in_lut,                        in_lut_size,                                 \
    		flt_lut,                       flt_lut_size,                                \
            chl_lut,                       chl_lut_size,                                \
            kloop_lut,                     kloop_lut_size,                              \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           (int *)bias

#define LUT_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                                                                  \
    		in_lut,                        in_lut_size,                                 \
    		flt_lut,                       flt_lut_size,                                \
            in_hw,                         out_hw,                                      \
            flt_hw,                        splitk,                                      \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            num_chl_per_grp,               num_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_prelu,          (const void *) fuse_param.prelu,             \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_elt_prelu,      (const void *) fuse_param.elt_prelu,         \
            leaky,                         elt_leaky,                                   \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8


#define IDX_KPARAM_LIST \
            pad_input,                                                                  \
            d_flt,                                                                      \
            conv_out,                                                                   \
            kloop_num,                      koff_num_pad,                               \
            in_hw,                         out_hw,                                      \
            flt_hw,                        out_nhw,                                     \
            conv_param.in_height,          conv_param.in_width,                         \
            conv_param.in_num,             conv_param.num_grp,                          \
            conv_param.num_chl,            num_chl_per_grp,                             \
            in_chl_per_grp_pad,            flt_chl_per_grp_pad,                         \
            conv_param.flt_height,         conv_param.flt_width,                        \
            num_flt_per_grp,               num_flt_per_grp_pad,                         \
            conv_param.out_height,         conv_param.out_width,                        \
            conv_param.stride_height,      conv_param.stride_width,                     \
            conv_param.pad_height,         conv_param.pad_width,                        \
            conv_param.hole_height,        conv_param.hole_width,                       \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_prelu,          (const void *) fuse_param.prelu,             \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_elt_prelu,      (const void *) fuse_param.elt_prelu,         \
            leaky,                         elt_leaky,                                   \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

#define MERGE_KPARAM_LIST \
        	conv_out,                      final_out,                                   \
        	spk_height_v1,                 spk_width_v8,                                \
        	out_hw,                        splitk * splitf,                             \
            conv_param.has_bias,           bias,                                        \
            fuse_param.has_activation,     clip_min,                                    \
            fuse_param.has_clip,           clip_max,                                    \
            fuse_param.has_prelu,          (const void *) fuse_param.prelu,             \
            fuse_param.has_elt,            (const int4 *) fuse_param.pre_data,          \
            fuse_param.has_elt_activation, elt_clip_min,                                \
            fuse_param.has_elt_clip,       elt_clip_max,                                \
            fuse_param.has_elt_prelu,      (const void *) fuse_param.elt_prelu,         \
            leaky,                         elt_leaky,                                   \
            fuse_param.has_concat,         concat_offset_v8,                            \
            concat_stride_v8

static std::vector<kernel_info_t> g_kernel_container;
static bool is_g_kernel_container_initialized = false;

static std::unordered_map<size_t, algo_param_t> g_conv_shape_hash;

void InitializeKernelContainer(std::vector<kernel_info_t> &g_kernel_container, ppl::common::datatype_t type)
{
    if( type == ppl::common::DATATYPE_FLOAT16 ) {
        Initialize2spkConvF1KernelContainer(g_kernel_container);
        Initialize2spkConvF3KernelContainer(g_kernel_container);
        Initialize2spkConvFNKernelContainer(g_kernel_container);
        Initialize2spkConvFSKernelContainer(g_kernel_container);
                      
        InitializeIdxnConvKernelContainer(g_kernel_container);
    }
    
    is_g_kernel_container_initialized = true;
}

__inline__ std::string GetConvShapeString(conv_param_t &conv_param)
{
    return std::string("b" + std::to_string(conv_param.in_num)  + \
                       "_c" + std::to_string(conv_param.num_chl) + \
                       "_d" + std::to_string(conv_param.num_flt) + \
                       "_g" + std::to_string(conv_param.num_grp) + \
                       "_h" + std::to_string(conv_param.in_height) + \
                       "_w" + std::to_string(conv_param.in_width) + \
                       "_r" + std::to_string(conv_param.flt_height) + \
                       "_s" + std::to_string(conv_param.flt_width) + \
                       "_p" + std::to_string(conv_param.pad_height) + \
                       "_q" + std::to_string(conv_param.pad_width) + \
                       "_u" + std::to_string(conv_param.stride_height) + \
                       "_v" + std::to_string(conv_param.stride_width) + \
                       "_y" + std::to_string(conv_param.hole_height) + \
                       "_x" + std::to_string(conv_param.hole_width) + \
                       "_");
}

__inline__ size_t GetConvShapeHashKey( conv_param_t &conv_param )
{
    return std::hash<std::string>{} (GetConvShapeString(conv_param));
}

uint64_t PPLCUDAConvolutionGetCompilationBufSize(ppl::common::datatype_t type, conv_param_t &conv_param, uint64_t workspace)
{
    int pad_size = GetPadSize(type);

    uint32_t num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    uint32_t num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    uint32_t num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    uint32_t num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size); 

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint32_t cvt_input_size = 0;
    uint32_t cvt_output_size = 0;

    if(is_in_grp_pad)
        cvt_input_size = GetCvtInputSize( type, conv_param, num_chl_per_grp_pad);

    if(is_out_grp_pad)
        cvt_output_size = getCvtOutputSize(type, conv_param, num_flt_per_grp_pad);

    uint32_t split_size = GetMaxSplitSize(type, conv_param, num_flt_per_grp_pad);

    uint64_t total_size = cvt_input_size + cvt_output_size + split_size;

    return total_size <= workspace ? total_size : workspace;
}
uint64_t PPLCUDAConvolutionGetRuntimeBufSize(
        ppl::common::datatype_t type,
        conv_param_t &conv_param,
        unsigned int splitk,
        unsigned int splitf,
        uint64_t workspace)
{
    int pad_size = GetPadSize(type);

    uint32_t num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    uint32_t num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    uint32_t num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    uint32_t num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size); 

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint32_t cvt_input_size = 0;
    uint32_t cvt_output_size = 0;

    if(is_in_grp_pad)
        cvt_input_size = GetCvtInputSize(type, conv_param, num_chl_per_grp_pad);
    if(is_out_grp_pad)
        cvt_output_size = getCvtOutputSize(type, conv_param, num_flt_per_grp_pad);

    uint32_t split_size = 0;
    
    if(splitk > 1 || splitf > 1)
        split_size = GetSplitKFSize(type, conv_param, num_flt_per_grp_pad, splitk, splitf);

    uint64_t total_size  = cvt_input_size + cvt_output_size + split_size;

    return total_size <= workspace ? total_size : workspace;
}

std::string ToString(int v) {
    std::stringstream ss;
    ss << v;
    return ss.str();
}

ppl::common::RetCode PPLCUDAConvolutionQuickSelectKernel(
        algo_param_t &algo_param,
        conv_param_t &conv_param) {
    int in_hw = conv_param.in_num * conv_param.in_height * conv_param.in_width;
    int out_hw = conv_param.in_num * conv_param.out_height * conv_param.out_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int chl_per_group = conv_param.num_chl / conv_param.num_grp;

    if(!is_g_kernel_container_initialized) 
        InitializeKernelContainer(g_kernel_container, ppl::common::DATATYPE_FLOAT16);

    if (algo_param.kid >= 0) {
        auto kid = algo_param.kid;
        algo_param.algo_name = g_kernel_container[kid].kname;
        algo_param.tiles.m_cta = g_kernel_container[kid].tile_m_per_cta;
        algo_param.tiles.m_warp = g_kernel_container[kid].tile_m_per_warp;
        algo_param.tiles.n_cta = g_kernel_container[kid].tile_n_per_cta;
        algo_param.tiles.n_warp = g_kernel_container[kid].tile_n_per_warp;
        algo_param.tiles.k_cta = g_kernel_container[kid].tile_k_per_cta;
        algo_param.tiles.k_per_step = g_kernel_container[kid].tile_k_per_step;
        algo_param.tiles.k_per_set = g_kernel_container[kid].tile_k_per_set;
        algo_param.tiles.flt_size = g_kernel_container[kid].flt_size;
        algo_param.tiles.flt_pad_size = g_kernel_container[kid].flt_pad_size;
        algo_param.tiles.cta_size_in_thd = g_kernel_container[kid].cta_size_in_thd;
    } else if (chl_per_group < 64) { // Use non-shared memory algo for small channel
        if (flt_hw > 9) {
            algo_param.tiles.m_cta = 128;
            algo_param.tiles.m_warp = 64;
        } else {
            algo_param.tiles.m_cta = 32;
            algo_param.tiles.m_warp = 16;
        }

        if (in_hw == out_hw) {
            algo_param.tiles.n_cta = 64;
            algo_param.tiles.n_warp = 32;
        } else {
            algo_param.tiles.n_cta = 32;
            algo_param.tiles.n_warp = 16;
        }

        if (conv_param.num_chl >= 16) {
            algo_param.tiles.k_cta = 32;
            algo_param.tiles.k_per_step = 32;
        } else {
            algo_param.tiles.k_cta = 16;
            algo_param.tiles.k_per_step = 16;
        }

        algo_param.tiles.cta_size_in_thd = (algo_param.tiles.m_cta / algo_param.tiles.m_warp) * \
                    (algo_param.tiles.n_cta / algo_param.tiles.n_warp) * \
                    WARP_SIZE;

        if(algo_param.tiles.k_per_step == 8)  algo_param.tiles.flt_pad_size = 2;
        else if(algo_param.tiles.k_per_step == 16) algo_param.tiles.flt_pad_size = 4;
        else if(algo_param.tiles.k_per_step == 32) algo_param.tiles.flt_pad_size = 8;

        algo_param.algo_name = "nvIdxnConv_hmma1688_nhwc_b"+ToString(algo_param.tiles.m_cta)+"x"+ToString(algo_param.tiles.n_cta)+
                                            "_w"+ToString(algo_param.tiles.m_warp)+"x"+ToString(algo_param.tiles.n_warp)+
                                            "_k"+ToString(algo_param.tiles.k_cta)+"_s"+ToString(algo_param.tiles.k_per_step)+"_nosmem";
    } else { // Use 3spk algo for large channel
        float min_pad = 1.0;
        algo_param.tiles.m_cta = 16;
        for (int32_t i = 128; i >= 16; i = i / 2) {
            if (out_hw < i) continue;
            float pad = 1.0 * (DivUp(out_hw, i) * i - out_hw) / out_hw;
            if (pad < min_pad)  {
                min_pad = pad;
                algo_param.tiles.m_cta = i;
            }
            if (min_pad < 0.1)  break;
        }

        algo_param.tiles.n_cta = 16;
        for (int32_t i = 128; i >= 16; i = i / 2) {
            int cout = conv_param.num_flt;
            if ((cout < 64 && i / cout == 1) || (cout >= 64 && cout % i == 0)) {
                algo_param.tiles.n_cta = i;
                break;
            }
        }

        if (conv_param.num_chl >= 128) {
            algo_param.tiles.k_cta = 64;
        } else {
            algo_param.tiles.k_cta = 32;
        }

        if (algo_param.tiles.m_cta == 128 && algo_param.tiles.n_cta == 128) {
            algo_param.tiles.m_cta = 64;
        }

        if (algo_param.tiles.m_cta * 4 < algo_param.tiles.n_cta) {
            algo_param.tiles.m_cta *= 2;
            algo_param.tiles.n_cta /= 2;
        }
        if (algo_param.tiles.n_cta *4 < algo_param.tiles.m_cta) {
            algo_param.tiles.m_cta /= 2;
            algo_param.tiles.n_cta *= 2;
        }

        algo_param.tiles.m_warp = algo_param.tiles.m_cta / 2;
        algo_param.tiles.n_warp = algo_param.tiles.n_cta / 2;
        algo_param.tiles.k_per_set = algo_param.tiles.k_cta / 2;
        if (algo_param.tiles.k_per_set <= 8) {
            algo_param.tiles.k_per_set = 16;
        }
        if (algo_param.tiles.m_warp <= 8) {
            algo_param.tiles.m_warp = 16;
        }
        if (algo_param.tiles.n_warp <= 8) {
            algo_param.tiles.n_warp = 16;
        }

        algo_param.tiles.cta_size_in_thd = (algo_param.tiles.m_cta / algo_param.tiles.m_warp) *  \
                               (algo_param.tiles.n_cta / algo_param.tiles.n_warp) *  \
                               (algo_param.tiles.k_cta / algo_param.tiles.k_per_set)  * \
                               WARP_SIZE;

        std::string f_size = "f1";
        algo_param.tiles.flt_size = 1;
        if (conv_param.flt_height == 3) {
            f_size = "f3";
            algo_param.tiles.flt_size = 3;
        } else if (conv_param.flt_height > 3) {
            f_size = "fn";
            algo_param.tiles.flt_size = 0;
        }
        algo_param.algo_name = "nv2spkConv_hmma1688_nhwc_"+f_size+"_b"+ToString(algo_param.tiles.m_cta)+"x"+ToString(algo_param.tiles.n_cta)+
                                                       "_w"+ToString(algo_param.tiles.m_warp)+"x"+ToString(algo_param.tiles.n_warp)+
                                                       "_k"+ToString(algo_param.tiles.k_cta)+"_s"+ToString(algo_param.tiles.k_per_set)+"_buf1";
    }
    return ppl::common::RC_SUCCESS;
}

string PPLCUDACompile(string name, string code, std::vector<const char*> compile_params, int device, bool include) {
    string ptx = ppl::nn::cuda::CUDANVRTCCompile(pair<string,string>(name, code), compile_params, device, include);
    return ptx;
}

float AlgoForwardTime(
    hipStream_t &stream, 
    string name,
    string code,
    std::vector<const char*> compile_params,
    int device,
    bool include,
    ppl::common::datatype_t type,
    int4* d_input,
    int4* d_flt,
    int4* d_output,
    int4* bias,
    int4* d_temp_buf, 
    algo_param_t &algo_param,
    conv_param_t &conv_param, 
    fuse_param_t &fuse_param,
    uint64_t workspace) 
{
    // printf("%s\n", name.c_str());
    string ptx = ppl::nn::cuda::CUDANVRTCCompile(pair<string,string>(name, code), compile_params, device, include);
    ppl::nn::cuda::CUDAModule* cuda_module = new ppl::nn::cuda::CUDAModule();
    cuda_module->SetSourceCode(name, ptx);
    hipFunction_t function = cuda_module->GetKernelFunc();

    int times = 4;
    float elapsed = 0;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    for (int i = 0; i < times; i++) {
        PPLCUDAConvolutionForwardJITImp( 
            stream, function, type, d_input, d_flt, d_output, bias, d_temp_buf,
            algo_param, conv_param, fuse_param);
    }
    hipEventRecord(end, stream);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed, begin, end);

    hipEventDestroy(begin);
    hipEventDestroy(end);
    delete cuda_module;
    return elapsed; 
}

ppl::common::RetCode PPLCUDAConvolutionSelectKernel(
        hipStream_t &stream, 
        ppl::common::datatype_t type,
        int4* d_input,
        int4* d_flt,
        int4* d_output,
        int4* bias,
        int4* d_temp_buf, 
        algo_param_t &algo_param,
        conv_param_t &conv_param, 
        fuse_param_t &fuse_param,
	    uint64_t workspace)
{
    if(!is_g_kernel_container_initialized)
        InitializeKernelContainer(g_kernel_container, type);

    size_t conv_shape_hash = GetConvShapeHashKey(conv_param);

    std::unordered_map<size_t, algo_param_t>::const_iterator conv_shape_hash_iterator = g_conv_shape_hash.find(conv_shape_hash);

    if(conv_shape_hash_iterator != g_conv_shape_hash.end()) {
        algo_param = conv_shape_hash_iterator->second;
        return ppl::common::RC_SUCCESS;
    }

    auto pre_algo_param = algo_param;
    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    
    int declare_times = 0;
    float minTime = FLT_MAX;
    float elapsed;

    const int SPLITK_OPTIONS[] = {1, 2, 4, 8};

    for(unsigned int spk = 0; spk < 1; spk++) {
        unsigned int splitk = SPLITK_OPTIONS[spk];

        for(unsigned int kid = 0; kid < g_kernel_container.size(); kid++) {
            unsigned int splitf = (g_kernel_container[kid].ktype == CONV_2SPK_FS) ? flt_hw : 1;
            printf("%d,%s\n", kid,g_kernel_container[kid].kname.c_str());
        
            if(!g_kernel_container[kid].CheckKernelTypeFeasible(conv_param.flt_height, conv_param.flt_width, num_chl_per_grp, splitk)) continue;

            if(!g_kernel_container[kid].CheckSplitkFeasible(num_chl_per_grp, splitk)) continue;

            if(!g_kernel_container[kid].CheckSplitfFeasible(splitf, splitk)) continue;

            algo_param_t temp_algo_param;
            temp_algo_param.kid = kid;
            temp_algo_param.splitk = splitk;
            temp_algo_param.splitf = splitf;
            temp_algo_param.algo_name = g_kernel_container[kid].kname;
            temp_algo_param.tiles.m_cta = g_kernel_container[kid].tile_m_per_cta;
            temp_algo_param.tiles.m_warp = g_kernel_container[kid].tile_m_per_warp;
            temp_algo_param.tiles.n_cta = g_kernel_container[kid].tile_n_per_cta;
            temp_algo_param.tiles.n_warp = g_kernel_container[kid].tile_n_per_warp;
            temp_algo_param.tiles.k_cta = g_kernel_container[kid].tile_k_per_cta;
            temp_algo_param.tiles.k_per_step = g_kernel_container[kid].tile_k_per_step;
            temp_algo_param.tiles.k_per_set = g_kernel_container[kid].tile_k_per_set;
            temp_algo_param.tiles.flt_size = g_kernel_container[kid].flt_size;
            temp_algo_param.tiles.flt_pad_size = g_kernel_container[kid].flt_pad_size;
            temp_algo_param.tiles.cta_size_in_thd = g_kernel_container[kid].cta_size_in_thd;

            if(!g_kernel_container[kid].CheckQuickSelectFeasible(pre_algo_param, conv_param.num_chl / conv_param.num_grp, splitk, splitf)) continue;

            std::string source = "";
            if (temp_algo_param.algo_name.find("Idxn") != std::string::npos) {
                GeneIdxnKernel(source, temp_algo_param.algo_name, 
                                       temp_algo_param.tiles.m_cta, 
                                       temp_algo_param.tiles.n_cta, 
                                       temp_algo_param.tiles.m_warp, 
                                       temp_algo_param.tiles.n_warp, 
                                       temp_algo_param.tiles.k_cta, 
                                       temp_algo_param.tiles.k_per_step, declare_times);
                declare_times++;
            } else {
                Gene2spkKernel(source, temp_algo_param.algo_name, 
                                       temp_algo_param.tiles.m_cta, 
                                       temp_algo_param.tiles.n_cta, 
                                       temp_algo_param.tiles.m_warp, 
                                       temp_algo_param.tiles.n_warp, 
                                       temp_algo_param.tiles.k_cta, 
                                       temp_algo_param.tiles.k_per_set, 
                                       temp_algo_param.splitk, 
                                       temp_algo_param.splitf, 
                                       1, declare_times);
                declare_times++;
            }

            std::vector<const char*> compile_params;
            elapsed = AlgoForwardTime(stream, 
                                      g_kernel_container[kid].kname,
                                      source,
                                      compile_params,
                                      0,
                                      true,
                                      type,
                                      d_input,
                                      d_flt,
                                      d_output,
                                      bias,
                                      d_temp_buf, 
                                      temp_algo_param,
                                      conv_param,
                                      fuse_param,
                                      workspace);
            
	        if(elapsed < minTime){
                algo_param = temp_algo_param;
	            minTime = elapsed;
	        }
        }
    }
    g_conv_shape_hash[conv_shape_hash] = algo_param;

    return ppl::common::RC_SUCCESS;
}

void PPLCUDAConvolutionForwardImp(
        hipStream_t &stream, 
        ppl::common::datatype_t type,
        int4* d_input,
        int4* d_flt,
        int4* d_output,
        int4* bias,
        int4* d_temp_buf,
        algo_param_t& algo_param,
        conv_param_t &conv_param,
        fuse_param_t &fuse_param)
{
    if(!is_g_kernel_container_initialized)
        InitializeKernelContainer(g_kernel_container, type);

    unsigned int kid = algo_param.kid;
    unsigned int splitk = algo_param.splitk;
    unsigned int splitf = algo_param.splitf;

    int pad_size = GetPadSize(type);

    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    int num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    int num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size);

    int in_hw  = conv_param.in_height * conv_param.in_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int out_hw = conv_param.out_height * conv_param.out_width;

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint64_t buf_off_v4 = 0;

    int4 *pad_input = d_input;
    int4 *pad_output = d_output;

    if(is_in_grp_pad) {
	    pad_input = d_temp_buf; 
	    buf_off_v4 += GetCvtInputSize(type, conv_param, num_chl_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);

        PPLCUDAConvolutionCvtInput(stream, pad_input, d_input, type, conv_param);
    }

    if(is_out_grp_pad) {
	    pad_output = d_temp_buf + buf_off_v4;
	    buf_off_v4 += getCvtOutputSize(type, conv_param, num_flt_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);
    } 

    int4 *final_out  = fuse_param.has_concat ? (int4 *) fuse_param.post_concat : pad_output;

    int4 *splitk_buf = d_temp_buf + buf_off_v4;
    int4 *conv_out   = (splitk > 1 || splitf > 1) ? splitk_buf : final_out;

    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    __half  leaky        = __float2half(fuse_param.leaky);
    __half  elt_leaky    = __float2half(fuse_param.elt_leaky);

    dim3 block_size, grid_size;

    block_size.x = g_kernel_container[kid].cta_size_in_thd;
    block_size.y = 1;
    block_size.z = 1;

    grid_size.x  = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, g_kernel_container[kid].tile_m_per_cta);
    grid_size.y  = DivUp(num_flt_per_grp_pad, g_kernel_container[kid].tile_n_per_cta);
    grid_size.z  = conv_param.num_grp * splitk * splitf;

    if(g_kernel_container[kid].ktype == CONV_IDXN_C2 || g_kernel_container[kid].ktype == CONV_IDXN_C4 || \
            g_kernel_container[kid].ktype == CONV_IDXN_C32) {
        int img_pad_size = pad_size;
        int flt_pad_size = g_kernel_container[kid].flt_pad_size;

        int out_nhw = out_hw * conv_param.in_num;

        int in_chl_per_grp_pad = Align(num_chl_per_grp, img_pad_size);
        int flt_chl_per_grp_pad = Align(num_chl_per_grp, flt_pad_size);
        int num_flt_per_grp_pad = Align(num_flt_per_grp, img_pad_size);

	    int kloop_num = DivUp(flt_hw * flt_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);
        int koff_num_pad = Align(kloop_num * (g_kernel_container[kid].tile_k_per_cta / flt_pad_size), WARP_SIZE);

        (g_kernel_container[kid].idx_kptr)<<<grid_size, block_size, 0, stream>>>(IDX_KPARAM_LIST);

    } else if(g_kernel_container[kid].ktype == CONV_2SPK_F1 || g_kernel_container[kid].ktype == CONV_2SPK_F3 || \
            g_kernel_container[kid].ktype == CONV_2SPK_FN || g_kernel_container[kid].ktype == CONV_2SPK_FS ) {

	    int kloop_num = (flt_hw / splitf) * DivUp(num_chl_per_grp_pad, g_kernel_container[kid].tile_k_per_cta);

        lut_t in_lut, flt_lut;
        int in_lut_size, flt_lut_size;
    
        InitializeInputLut(in_lut_size, in_lut.idx, conv_param.flt_height, conv_param.flt_width, conv_param.in_height,
                conv_param.in_width, conv_param.pad_height, conv_param.pad_width, conv_param.hole_height, conv_param.hole_width,
                num_chl_per_grp_pad, conv_param.num_grp, g_kernel_container[kid].tile_k_per_cta, pad_size);

        InitializeFilterLut(flt_lut_size, flt_lut.idx, conv_param.flt_height, conv_param.flt_width, num_chl_per_grp_pad,
                g_kernel_container[kid].tile_k_per_cta, pad_size);

        if(splitk == 1) {
            (g_kernel_container[kid].lut_kptr)<<<grid_size, block_size, 0, stream>>>(LUT_KPARAM_LIST);
        } else {
            int chl_lut_size, kloop_lut_size;
            struct chl_lut_t chl_lut;
            struct kloop_lut_t kloop_lut;

            InitializeChlLut(chl_lut_size, chl_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                    g_kernel_container[kid].tile_k_per_cta, splitk);
            InitializeKloopLut(kloop_lut_size, kloop_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                    g_kernel_container[kid].tile_k_per_cta, splitk, splitf, flt_hw);

            (g_kernel_container[kid].spk_kptr)<<<grid_size, block_size, 0, stream>>>(SPK_KPARAM_LIST);
        }
    }
    
    if(splitk > 1 || splitf > 1) {
        int spk_width_v8   = num_flt_per_grp_pad * conv_param.num_grp / pad_size;
        int spk_height_v1  = out_hw * conv_param.in_num;

        dim3 merge_grid_size, merge_block_size;
        merge_block_size.x = 64;
        merge_block_size.y = 1;
        merge_block_size.z = 1;

        merge_grid_size.x  = spk_height_v1;
        merge_grid_size.y  = DivUp(spk_width_v8, merge_block_size.x);
        merge_grid_size.z  = 1;

        MergeConvSplitResults<<<merge_grid_size, merge_block_size, 0, stream>>>(MERGE_KPARAM_LIST);
    }

    if(is_out_grp_pad) {
        PPLCUDAConvolutionCvtOutput(stream, d_output, final_out, type, conv_param);
    }

}

#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      exit(1);                                                    \
    }                                                             \
  } while(0)
#define CUDA_SAFE_CALL(x)                                         \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      std::cerr << "\nerror: " #x " failed with error "           \
                << msg << '\n';                                   \
      exit(1);                                                    \
    }                                                             \
  } while(0)

#define CUDA_RUNTIME_CALL(x)                                    \
  do {                                                            \
    hipError_t result = x;                                       \
    if (result != hipSuccess) {                                 \
      const char *msg = hipGetErrorName(result);                   \
      std::cerr << "\nerror: " #x " failed with error "           \
                << msg << '\n';                                   \
      exit(1);                                                    \
    }                                                             \
  } while(0)

void PPLCUDAConvolutionForwardJITImp(
    hipStream_t &stream,
    hipFunction_t function,
    ppl::common::datatype_t type,
    int4* d_input,
    int4* d_flt,
    int4* d_output,
    int4* bias,
    int4* d_temp_buf,
    algo_param_t &algo_param,
    conv_param_t &conv_param,
    fuse_param_t &fuse_param)
{
    unsigned int kid = algo_param.kid;
    unsigned int splitk = algo_param.splitk;
    unsigned int splitf = algo_param.splitf;

    int pad_size = GetPadSize(type);

    int num_chl_per_grp = conv_param.num_chl / conv_param.num_grp;
    int num_flt_per_grp = conv_param.num_flt / conv_param.num_grp;

    int num_chl_per_grp_pad = Align(num_chl_per_grp, pad_size);
    int num_flt_per_grp_pad = Align(num_flt_per_grp, pad_size);

    int in_hw  = conv_param.in_height * conv_param.in_width;
    int flt_hw = conv_param.flt_height * conv_param.flt_width;
    int out_hw = conv_param.out_height * conv_param.out_width;

    int concat_offset_v8 = fuse_param.concat_offset / pad_size;
    int concat_stride_v8 = fuse_param.concat_stride / pad_size;

    bool  is_in_grp_pad = num_chl_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;
    bool is_out_grp_pad = num_flt_per_grp_pad != num_chl_per_grp && conv_param.num_grp != 1;

    uint64_t buf_off_v4 = 0;

    int4 *pad_input = d_input;
    int4 *pad_output = d_output;

    if(is_in_grp_pad) {
	    pad_input = d_temp_buf; 
	    buf_off_v4 += GetCvtInputSize(type, conv_param, num_chl_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);

        PPLCUDAConvolutionCvtInput(stream, pad_input, d_input, type, conv_param);
    }

    if(is_out_grp_pad) {
	    pad_output = d_temp_buf + buf_off_v4;
	    buf_off_v4 += getCvtOutputSize(type, conv_param, num_flt_per_grp_pad) / (_4INT_TO_INT4_ * _INT_TO_4BYTE_);
    } 

    int4 *final_out  = fuse_param.has_concat ? (int4 *) fuse_param.post_concat : pad_output;

    int4 *splitk_buf = d_temp_buf + buf_off_v4;
    int4 *conv_out   = (splitk > 1 || splitf > 1) ? splitk_buf : final_out;

    __half2 clip_min     = __float2half2_rn(fuse_param.clip_min);
    __half2 clip_max     = __float2half2_rn(fuse_param.clip_max);
    __half2 elt_clip_min = __float2half2_rn(fuse_param.elt_clip_min);
    __half2 elt_clip_max = __float2half2_rn(fuse_param.elt_clip_max);
    __half  leaky        = __float2half(fuse_param.leaky);
    __half  elt_leaky    = __float2half(fuse_param.elt_leaky);
    
    int tile_n = algo_param.tiles.n_cta;
    int tile_m = algo_param.tiles.m_cta;
    int cta_k = algo_param.tiles.k_cta;

    dim3 block_size, grid_size;
    block_size.x = algo_param.tiles.cta_size_in_thd;;
    block_size.y = 1;
    block_size.z = 1;

    grid_size.x  = DivUp(conv_param.in_num * conv_param.out_height * conv_param.out_width, tile_m);
    grid_size.y  = DivUp(num_flt_per_grp_pad, tile_n);
    grid_size.z  = conv_param.num_grp * splitk * splitf;

    // int has_relu = fuse_param.has_activation == 1? 1:0;
    // int has_elt_relu = fuse_param.has_elt_activation == 1 ? 1 : 0;
    const int4* pre_data = (const int4*)fuse_param.pre_data;
    const void* prelu = (const void*)fuse_param.prelu;
    const void* elt_prelu = (const void*)fuse_param.elt_prelu;


    if (algo_param.algo_name.find("Idxn") != std::string::npos) {
        int img_pad_size = pad_size;
        int flt_pad_size = algo_param.tiles.flt_pad_size;

        int out_nhw = out_hw * conv_param.in_num;

        int in_chl_per_grp_pad = Align(num_chl_per_grp, img_pad_size);
        int flt_chl_per_grp_pad = Align(num_chl_per_grp, flt_pad_size);
        int num_flt_per_grp_pad = Align(num_flt_per_grp, img_pad_size);

	    int kloop_num = DivUp(flt_hw * flt_chl_per_grp_pad, cta_k);
        int koff_num_pad = Align(kloop_num * (cta_k / flt_pad_size), WARP_SIZE);
        
        void *args[] = {&pad_input, &d_flt, &conv_out, 
                        &kloop_num, &koff_num_pad, &in_hw, &out_hw,
                        &flt_hw, &out_nhw, &conv_param.in_height, &conv_param.in_width,
                        &conv_param.in_num, &conv_param.num_grp, &conv_param.num_chl, &num_chl_per_grp,
                        &in_chl_per_grp_pad, &flt_chl_per_grp_pad,
                        &conv_param.flt_height, &conv_param.flt_width, &num_flt_per_grp, &num_flt_per_grp_pad,
                        &conv_param.out_height, &conv_param.out_width, &conv_param.stride_height, &conv_param.stride_width,
                        &conv_param.pad_height, &conv_param.pad_width, &conv_param.hole_height, &conv_param.hole_width,
                        &conv_param.has_bias, &bias, &fuse_param.has_activation, &clip_min,
                        &fuse_param.has_clip, &clip_max, 
                        &fuse_param.has_prelu, &prelu,
                        &fuse_param.has_elt, &(pre_data),
                        &fuse_param.has_elt_activation, &elt_clip_min, &fuse_param.has_elt_clip, &elt_clip_max,
                        &fuse_param.has_elt_prelu, &(elt_prelu), &leaky, &elt_leaky,
                        &fuse_param.has_concat, &concat_offset_v8, &concat_stride_v8};

        CUDA_SAFE_CALL(hipModuleLaunchKernel(function, grid_size.x, grid_size.y, grid_size.z, 
                        block_size.x, block_size.y, block_size.z,
                        0, stream, args, 0));    
    } else if (algo_param.algo_name.find("2spk") != std::string::npos) {

        // std::cout << "block size " << block_size.x << std::endl;
        // std::cout << "grid_size " << grid_size.x << " " << grid_size.y << " " << grid_size.z << std::endl;
        int kloop_num = (flt_hw / splitf) * DivUp(num_chl_per_grp_pad, cta_k);//g_kernel_container[kid].tile_k_per_cta);

        lut_t in_lut, flt_lut;
        int in_lut_size, flt_lut_size;

        InitializeInputLut(in_lut_size, in_lut.idx, conv_param.flt_height, conv_param.flt_width, conv_param.in_height,
                conv_param.in_width, conv_param.pad_height, conv_param.pad_width, conv_param.hole_height, conv_param.hole_width,
                num_chl_per_grp_pad, conv_param.num_grp, cta_k, pad_size);

        InitializeFilterLut(flt_lut_size, flt_lut.idx, conv_param.flt_height, conv_param.flt_width, num_chl_per_grp_pad,
            cta_k, pad_size);
        if (splitk == 1) {
            void *args[] = {&pad_input, &d_flt, &conv_out, &kloop_num,
                        &in_lut, &in_lut_size, &flt_lut, &flt_lut_size, &in_hw, &out_hw,
                        &flt_hw, &splitk, &conv_param.in_height, &conv_param.in_width,
                        &conv_param.in_num, &conv_param.num_grp, &num_chl_per_grp, &num_chl_per_grp_pad,
                        &conv_param.flt_height, &conv_param.flt_width, &num_flt_per_grp, &num_flt_per_grp_pad,
                        &conv_param.out_height, &conv_param.out_width, &conv_param.stride_height, &conv_param.stride_width,
                        &conv_param.pad_height, &conv_param.pad_width, &conv_param.hole_height, &conv_param.hole_width,
                        &conv_param.has_bias, &bias, &fuse_param.has_activation, &clip_min,
                        &fuse_param.has_clip, &clip_max, 
                        &fuse_param.has_prelu, &prelu,
                        &fuse_param.has_elt, &(pre_data),
                        &fuse_param.has_elt_activation, &elt_clip_min, &fuse_param.has_elt_clip, &elt_clip_max,
                        &fuse_param.has_elt_prelu, &(elt_prelu), &leaky, &elt_leaky,
                        &fuse_param.has_concat, &concat_offset_v8, &concat_stride_v8};
            CUDA_SAFE_CALL(hipModuleLaunchKernel(function, grid_size.x, grid_size.y, grid_size.z, 
                        block_size.x, block_size.y, block_size.z,
                        0, stream, args, 0));
        } else {
            int chl_lut_size, kloop_lut_size;
            struct chl_lut_t chl_lut;
            struct kloop_lut_t kloop_lut;

            InitializeChlLut(chl_lut_size, chl_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                    g_kernel_container[kid].tile_k_per_cta, splitk);
            InitializeKloopLut(kloop_lut_size, kloop_lut.idx, conv_param.num_chl, conv_param.num_grp, pad_size,
                    g_kernel_container[kid].tile_k_per_cta, splitk, splitf, flt_hw);
            
            void* args[] = {&pad_input, &d_flt, &conv_out, &kloop_num,
                &in_lut, &in_lut_size, &flt_lut, &flt_lut_size,
                &chl_lut, &chl_lut_size, &kloop_lut, &kloop_lut_size,
                &in_hw, &out_hw, &flt_hw, &splitk,
                &conv_param.in_height, &conv_param.in_width,
                &conv_param.in_num, &conv_param.num_grp, &num_chl_per_grp, &num_chl_per_grp_pad,
                &conv_param.flt_height, &conv_param.flt_width, &num_flt_per_grp, &num_flt_per_grp_pad,
                &conv_param.out_height, &conv_param.out_width, &conv_param.stride_height, &conv_param.stride_width,
                &conv_param.pad_height, &conv_param.pad_width, &conv_param.hole_height, &conv_param.hole_width,
                &conv_param.has_bias, &bias };
            CUDA_SAFE_CALL(hipModuleLaunchKernel(function, grid_size.x, grid_size.y, grid_size.z, 
                block_size.x, block_size.y, block_size.z,
                0, stream, args, 0));
        }
    } 
    else {

    }
    
    if(splitk > 1 || splitf > 1) {
        int spk_width_v8   = num_flt_per_grp_pad * conv_param.num_grp / pad_size;
        int spk_height_v1  = out_hw * conv_param.in_num;

        dim3 merge_grid_size, merge_block_size;
        merge_block_size.x = 64;
        merge_block_size.y = 1;
        merge_block_size.z = 1;

        merge_grid_size.x  = spk_height_v1;
        merge_grid_size.y  = DivUp(spk_width_v8, merge_block_size.x);
        merge_grid_size.z  = 1;

        MergeConvSplitResults<<<merge_grid_size, merge_block_size, 0, stream>>>(MERGE_KPARAM_LIST);
    }
    if(is_out_grp_pad) {
        PPLCUDAConvolutionCvtOutput(stream, d_output, final_out, type, conv_param);
    }

}